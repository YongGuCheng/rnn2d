#include "hip/hip_runtime.h"
#include <rnn2d/lstm_gpu.h>

#include <algorithm>
#include <cmath>
#include <cassert>
#include <cstdio>

#include <glog/logging.h>
#include <thrust/device_vector.h>

#include <rnn2d/activation.h>
#include <rnn2d/cuda_utils.h>
#include <rnn2d/math_gpu.h>

#define BLOCK_SIZE 256
#define GRID_SIZE 128

#define STREAMS_CREATE(N)                               \
  for (int i = 0; i < (N); ++i) {                       \
    CHECK_CUDA_CALL(hipStreamCreate(&stream[i]));      \
  }

#define STREAMS_DESTROY(N)                              \
  for (int i = 0; i < (N); ++i) {                       \
    CHECK_CUDA_CALL(hipStreamDestroy(stream[i]));      \
  }

#define STREAMS_SYNCHRONIZE(N)                                  \
  for (int i = 0; i < (N); ++i) {                               \
    CHECK_CUDA_CALL(hipStreamSynchronize(stream[i]));          \
  }

// Useful defines to index several arrays:
// Input array
#define I_ptr(y, x, n, d)                       \
  (I  + (((y) * W + (x)) * N + (n)) * K + (d))
#define dI_ptr(y, x, n, d)                      \
  (dI + (((y) * W + (x)) * N + (n)) * K + (d))
// Output array
#define O_ptr(y, x, n, z, d)                                    \
  (O  + ((((y) * W + (x)) * N + (n)) * 4 + (z)) * D + (d))
#define dO_ptr(y, x, n, z, d)                                   \
  (dO + ((((y) * W + (x)) * N + (n)) * 4 + (z)) * D + (d))
// Bias array
#define B_ptr(z, g, d)                                  \
  (P  + (z) * (1 + K + D + D) * 5 * D + (g) * D + (d))
#define dB_ptr(z, g, d)                                 \
  (dP + (z) * (1 + K + D + D) * 5 * D + (g) * D + (d))
// Input weights array
#define W_ptr(z, k, g, d)                                               \
  (P  + ((z) * (1 + K + D + D) + 1 + (k)) * 5 * D + (g) * D + (d))
#define dW_ptr(z, k, g, d)                                              \
  (dP + ((z) * (1 + K + D + D) + 1 + (k)) * 5 * D + (g) * D + (d))
// Recurrent-y array
#define U_ptr(z, d1, g, d2)                                             \
  (P  + ((z) * (1 + K + D + D) + 1 + K + (d1)) * 5 * D + (g) * D + (d2))
#define dU_ptr(z, d1, g, d2)                                            \
  (dP + ((z) * (1 + K + D + D) + 1 + K + (d1)) * 5 * D + (g) * D + (d2))
// Recurrent-x array
#define V_ptr(z, d1, g, d2)                                             \
  (P  + ((z) * (1 + K + D + D) + 1 + K + D + (d1)) * 5 * D + (g) * D + (d2))
#define dV_ptr(z, d1, g, d2)                                            \
  (dP + ((z) * (1 + K + D + D) + 1 + K + D + (d1)) * 5 * D + (g) * D + (d2))

// Reserve array
#define Q_ptr(z, y, x, n, g, d)                                         \
  (Q  + (((((z) * H + (y)) * W + (x)) * N + (n)) * 5 + (g)) * D + (d))
// Workspace array
#define Z_ptr(g, z, y, x, n, d)                                         \
  (Z  + (((((g) * 4 + (z)) * H + (y)) * W + (x)) * N + (n)) * D + (d))

template <typename T>
inline size_t get_inference_workspace_size(
    const int H, const int W, const int N, const int D) {
  const size_t tmpd_size = 4 * H * W * N * 5 * D * sizeof(T);
  const size_t ptrs_size = 2 * 3 * 4 * std::min(H, W) * sizeof(T*);
  return tmpd_size + ptrs_size;
}

template <typename T>
inline size_t get_training_workspace_size(
    const int H, const int W, const int N, const int D) {
  const size_t tmpd_size = 3 * 4 * H * W * N * D * sizeof(T);
  const size_t ptrs_size = 2 * 3 * 4 * std::min(H, W) * sizeof(T*);
  return tmpd_size + ptrs_size;
}

template <typename T>
inline size_t get_training_reserve_size(
    const int H, const int W, const int N, const int D) {
  return 4 * H * W * N * 5 * D * sizeof(T);
}

template <typename T>
__global__
void kernel_fill1D(const int n, T* x, const T v) {
  for (int i = thGx; i < n; i += NTGx) {
    x[i] = v;
  }
}

template <typename T>
__global__
void kernel_init_Q_with_bias(
    const int H, const int W, const int N, const int K, const int D,
    const T* P, T* Q) {
  for (int ii = thGi; ii < 4 * H * W * N * 5 * D; ii += NTG) {
    const int d = ii % D;                      // d \in [0 ... D-1]
    const int g = (ii / D) % 5;                // g \in [0 ... 5]
    const int n = (ii / (5 * D)) % N;          // n \in [0 ... N-1]
    const int x = (ii / (N * 5 * D)) % W;      // x \in [0 ... W-1]
    const int y = (ii / (W * N * 5 * D)) % H;  // y \in [0 ... H-1]
    const int z = (ii / (H * W * N * 5 * D));  // z \in [0 ... 3]
    *Q_ptr(z, y, x, n, g, d) = *B_ptr(z, g, d);
  }
}

template <typename T, typename FG, typename FI, typename FO>
__global__
void kernel_fw_elemwise_ops(const int H, const int W, const int N, const int D,
                            const int t, const int Tn, const int Tmin,
                            const int* S, T* Q, T* O) {
  for (int ii = thGi; ii < 4 * Tn * N * D; ii += NTG) {
    const int d = ii % D;                // dimension
    const int n = (ii / D) % N;          // batch sample
    const int e = (ii / (N * D)) % Tn;   // element in diagonal
    const int z = (ii / (Tn * N * D));   // direction
    const int i = e + Tmin;
    const int j = t - i;
    const int y  = (z == 0 || z == 1) ? i : H - i - 1;
    const int x  = (z == 0 || z == 2) ? j : W - j - 1;
    if (S == nullptr || (y < S[n * 2] && x < S[n * 2 + 1])) {
      const int yp = (z == 0 || z == 1) ? y - 1 : y + 1;
      const int xp = (z == 0 || z == 2) ? x - 1 : x + 1;
      const T f_a   = FI::f(*Q_ptr(z, y, x, n, 0, d));  // f_i(input)
      const T f_gi  = FG::f(*Q_ptr(z, y, x, n, 1, d));  // f_g(input gate)
      const T f_go  = FG::f(*Q_ptr(z, y, x, n, 2, d));  // f_g(output gate)
      const T f_gfy = FG::f(*Q_ptr(z, y, x, n, 3, d));  // f_g(forget_y gate)
      const T f_gfx = FG::f(*Q_ptr(z, y, x, n, 4, d));  // f_g(forget_x gate)
      const T C_10  = (yp >= 0 && yp < H) ? *Q_ptr(z, yp, x, n, 0, d) : 0;
      const T C_01  = (xp >= 0 && xp < W) ? *Q_ptr(z, y, xp, n, 0, d) : 0;
      const T C_00  = f_gi * f_a + f_gfy * C_10 + f_gfx * C_01;  // state
      const T O_00  = f_go * FO::f(C_00);                        // output
      *Q_ptr(z, y, x, n, 0, d) = C_00;
      *Q_ptr(z, y, x, n, 1, d) = f_gi;
      *Q_ptr(z, y, x, n, 2, d) = f_go;
      *Q_ptr(z, y, x, n, 3, d) = f_gfy;
      *Q_ptr(z, y, x, n, 4, d) = f_gfx;
      *O_ptr(y, x, n, z, d)    = O_00;
    } else {
      *Q_ptr(z, y, x, n, 0, d) = 0;
      *Q_ptr(z, y, x, n, 1, d) = 0;
      *Q_ptr(z, y, x, n, 2, d) = 0;
      *Q_ptr(z, y, x, n, 3, d) = 0;
      *Q_ptr(z, y, x, n, 4, d) = 0;
      *O_ptr(y, x, n, z, d)    = 0;
    }
  }
}

/* 2D-LSTM forward pass running on the GPU
 * H -> maximum height
 * W -> maximum width
 * N -> batch size
 * K -> input dimensions/channels
 * D -> output dimensions/channels
 * I -> input data (layout: H x W x N x K)
 * S -> input sizes (height and width of each sample, layout: N x 2)
 * P -> parameters (size: 4 * (1 + K + D + D) * 5 * D)
 * O -> output data (layout: H x W x N x 4 x D)
 * Q -> gates pre-activations and cells (layout: 4 x H x W x N x 5 x D)
 */
template <typename T, typename FG, typename FI, typename FO>
inline void fw_training(
    const int H, const int W, const int N, const int K, const int D,
    const T* I, const int* S, const T* P, T* O, void* wspace, void* rspace) {
  CHECK_NOTNULL(I);
  CHECK_NOTNULL(P);
  CHECK_NOTNULL(O);
  CHECK_NOTNULL(wspace);
  // Prepare cublas handler and streams
  hipblasHandle_t handle;
  CHECK_CUBLAS_CALL(hipblasCreate(&handle));


  T* Q = reinterpret_cast<T*>(rspace != nullptr ? rspace : wspace);

  // Initialize gates with bias
  // [A,Gi,Go,Gx,Gy](x,y) = [b_a,b_i,b_o,b_x,b_y]
  kernel_init_Q_with_bias<T><<<GRID_SIZE, BLOCK_SIZE>>>(H, W, N, K, D, P, Q);
  CHECK_LAST_CUDA_CALL();


  const T** ptrs_gpu = reinterpret_cast<const T**>(
      (char*)wspace + sizeof(T) * ((rspace != nullptr)
                                   // workspace during training
                                   ? (3 * 4 * H * W * N * D)
                                   // workspace during inference
                                   : (4 * H * W * N * 5 * D)));
  const T** ptrs_cpu = nullptr;
  CHECK_CUDA_CALL(hipHostMalloc(
      &ptrs_cpu, sizeof(const T**) * 2 * 3 * 4 * std::min(H, W)));

  // Multiply inputs by weights:
  // [A,Gi,Go,Gx,Gy](x,y) += I(x,y) * [W_a,W_i,W_o,W_x,W_y]
  {
    ptrs_cpu[0]  = ptrs_cpu[1] = ptrs_cpu[2] = ptrs_cpu[3] = I;
    ptrs_cpu[4]  = W_ptr(0, 0, 0, 0); ptrs_cpu[5] = W_ptr(1, 0, 0, 0);
    ptrs_cpu[6]  = W_ptr(2, 0, 0, 0); ptrs_cpu[7] = W_ptr(3, 0, 0, 0);
    ptrs_cpu[8]  = Q_ptr(0, 0, 0, 0, 0, 0);
    ptrs_cpu[9]  = Q_ptr(1, 0, 0, 0, 0, 0);
    ptrs_cpu[10] = Q_ptr(2, 0, 0, 0, 0, 0);
    ptrs_cpu[11] = Q_ptr(3, 0, 0, 0, 0, 0);
    CHECK_CUDA_CALL(hipMemcpy(
        ptrs_gpu, ptrs_cpu, 12 * sizeof(const T**), hipMemcpyHostToDevice));
    CHECK_CUBLAS_CALL(gemm_gpu_batched<T>(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, H * W * N, 5 * D, K,
        1.0, ptrs_gpu, K,
        ptrs_gpu + 4, 5 * D,
        1.0, const_cast<T**>(ptrs_gpu) + 8, 5 * D, 4));
  }

  // Process the image diagonal-wise (there are H + W - 1 diagonals to process)
  {
    const T** ptrs_cpu_x = ptrs_cpu + 0 * 3 * 4 * std::min(H, W);
    const T** ptrs_cpu_y = ptrs_cpu + 1 * 3 * 4 * std::min(H, W);

    for (int t = 0; t < H + W - 1; ++t) {
      // Compute number of elements in the diagonal
      const int Tmin = std::max(0, t - W + 1);
      const int Tmax = std::min(t, H - 1);
      const int Tn   = (Tmax - Tmin) + 1;

      // Matrix multiplications to compute the input to the gates from the
      // recurrent connections.
      // [A,Gi,Go,Gx,Gy](x,y) += O(x,y-1) * [U_a,U_i,U_o,U_x,U_y]
      // [A,Gi,Go,Gx,Gy](x,y) += O(x-1,y) * [V_a,V_i,V_o,V_x,V_y]
      int batch_mul_size_x = 0, batch_mul_size_y = 0;
      for (int z = 0; z < 4; ++z) {
        for (int e = 0; e < Tn; ++e) {
          // (y, x) coordinates of the e-th element in the z-th diagonal.
          const int i = e + Tmin;
          const int j = t - i;
          const int y  = (z == 0 || z == 1) ? i : H - i - 1;
          const int x  = (z == 0 || z == 2) ? j : W - j - 1;
          const int yp = (z == 0 || z == 1) ? y - 1 : y + 1;
          const int xp = (z == 0 || z == 2) ? x - 1 : x + 1;
          if (yp >= 0 && yp <= H - 1) {
            ptrs_cpu_y[batch_mul_size_y + 0 * 4 * std::min(H, W)] =
                O_ptr(yp, x, 0, z, 0);
            ptrs_cpu_y[batch_mul_size_y + 1 * 4 * std::min(H, W)] =
                U_ptr(z, 0, 0, 0);
            ptrs_cpu_y[batch_mul_size_y + 2 * 4 * std::min(H, W)] =
                Q_ptr(z, y, x, 0, 0, 0);
            ++batch_mul_size_y;
          }
          if (xp >= 0 && xp <= W - 1) {
            ptrs_cpu_x[batch_mul_size_x + 0 * 4 * std::min(H, W)] =
                O_ptr(y, xp, 0, z, 0);
            ptrs_cpu_x[batch_mul_size_x + 1 * 4 * std::min(H, W)] =
                V_ptr(z, 0, 0, 0);
            ptrs_cpu_x[batch_mul_size_x + 2 * 4 * std::min(H, W)] =
                Q_ptr(z, y, x, 0, 0, 0);
            ++batch_mul_size_x;
          }
        }
      }
      // Copy pointers to the gpu for batched multiplications
      CHECK_CUDA_CALL(
          hipMemcpy(ptrs_gpu, ptrs_cpu,
                     sizeof(const T**) * 2 * 3 * 4 * std::min(H, W),
                     hipMemcpyHostToDevice));
      // [A,Gi,Go,Gx,Gy](x,y) += O(x-1,y) * [V_a,V_i,V_o,V_x,V_y]
      const T** Ox_ptrs = ptrs_gpu + 0 * 4 * std::min(H, W);
      const T** V_ptrs = ptrs_gpu + 1 * 4 * std::min(H, W);
      T** Qx_ptrs = const_cast<T**>(ptrs_gpu) + 2 * 4 * std::min(H, W);
      CHECK_CUBLAS_CALL(
          gemm_gpu_batched<T>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, 5 * D, D,
                              1.0, Ox_ptrs, 4 * D, V_ptrs, 5 * D,
                              1.0, Qx_ptrs, 5 * D, batch_mul_size_x));
      // [A,Gi,Go,Gx,Gy](x,y) += O(x,y-1) * [U_a,U_i,U_o,U_x,U_y]
      const T** Oy_ptrs = ptrs_gpu + (3 + 0) * 4 * std::min(H, W);
      const T** U_ptrs = ptrs_gpu + (3 + 1) * 4 * std::min(H, W);
      T** Qy_ptrs = const_cast<T**>(ptrs_gpu + (3 + 2) * 4 * std::min(H, W));
      CHECK_CUBLAS_CALL(
          gemm_gpu_batched<T>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, 5 * D, D,
                              1.0, Oy_ptrs, 4 * D, U_ptrs, 5 * D,
                              1.0, Qy_ptrs, 5 * D, batch_mul_size_y));

      kernel_fw_elemwise_ops<T, FG, FI, FO>
          <<<GRID_SIZE, BLOCK_SIZE>>>(H, W, N, D, t, Tn, Tmin, S, Q, O);
      CHECK_LAST_CUDA_CALL();
    }
  }

  CHECK_CUBLAS_CALL(hipblasDestroy(handle));
  CHECK_CUDA_CALL(hipHostFree(ptrs_cpu));
}

template <typename T, typename FG, typename FI, typename FO>
__global__
void kernel_bw_elemwise_ops(const int H, const int W, const int N, const int D,
                            const int t, const int Tn, const int Tmin,
                            const int* S, T* Q, T* Z) {
  for (int ii = thGi; ii < 4 * Tn * N * D; ii += NTG) {
    const int d = ii % D;
    const int n = (ii / D) % N;
    const int e = (ii / (N * D)) % Tn;
    const int z = (ii / (Tn * N * D));
    const int i = e + Tmin;
    const int j = t - i;
    const int y = (z == 0 || z == 1) ? i : H - i - 1;
    const int x = (z == 0 || z == 2) ? j : W - j - 1;
    T* dA_00   = Q_ptr(z, y, x, n, 0, d);   // currently contains C_00
    T* dGi_00  = Q_ptr(z, y, x, n, 1, d);   // currenlty contains f(Gi_00)
    T* dGo_00  = Q_ptr(z, y, x, n, 2, d);   // currently contains f(Go_00)
    T* dGfy_00 = Q_ptr(z, y, x, n, 3, d);   // currently contains f(Gfy_00)
    T* dGfx_00 = Q_ptr(z, y, x, n, 4, d);   // currently contains f(Gfx_00)
    if (S == nullptr || (y < S[n * 2] && x < S[n * 2 + 1])) {
      const int yn = (z == 0 || z == 1) ? y + 1 : y - 1;  // next y
      const int xn = (z == 0 || z == 2) ? x + 1 : x - 1;  // next x
      const int yp = (z == 0 || z == 1) ? y - 1 : y + 1;  // previous y
      const int xp = (z == 0 || z == 2) ? x - 1 : x + 1;  // previous x
      const T C_00    = *dA_00;
      const T fGi_00  = *dGi_00;
      const T fGo_00  = *dGo_00;
      const T fGfy_00 = *dGfy_00;
      const T fGfx_00 = *dGfx_00;
      const T dO_00   = *Z_ptr(0, z, y, x, n, d);
      const T C_10    = (yp >= 0 && yp < H) ? *Q_ptr(z, yp, x, n, 0, d) : 0;
      const T C_01    = (xp >= 0 && xp < W) ? *Q_ptr(z, y, xp, n, 0, d) : 0;
      const T fA_00   = fGi_00 != 0.0 ?
          (C_00 - C_10 * fGfy_00 - C_01 * fGfx_00) / fGi_00 : 0.0;
      // Z_10 = dC(y+1, x) * f(Gfy(y+1, x))
      const T Z_10  = (yn >= 0 && yn < H) ? *Z_ptr(1, z, yn, x, n, d) : 0;
      // Z_01 = dC(y, x+1) * f(Gfx(y, x+1))
      const T Z_01  = (xn >= 0 && xn < W) ? *Z_ptr(2, z, y, xn, n, d) : 0;
      const T dC_00  = dO_00 * FO::df(C_00) * fGo_00 + Z_10 + Z_01;
      *dGo_00 = dO_00 * FO::f(C_00) * FG::df2(fGo_00);
      *dGfy_00 = (yp >= 0 && yp < H) ? (dC_00) * C_10 * FG::df2(fGfy_00) : 0;
      *dGfx_00 = (xp >= 0 && xp < W) ? (dC_00) * C_01 * FG::df2(fGfx_00) : 0;
      *dGi_00  = (dC_00) * fA_00 * FG::df2(fGi_00);
      *dA_00   = (dC_00) * FI::df2(fA_00) * fGi_00;
      *Z_ptr(1, z, y, x, n, d) = dC_00 * fGfy_00;
      *Z_ptr(2, z, y, x, n, d) = dC_00 * fGfx_00;
    } else {
      *dA_00   = 0;
      *dGi_00  = 0;
      *dGo_00  = 0;
      *dGfy_00 = 0;
      *dGfx_00 = 0;
      *Z_ptr(1, z, y, x, n, d) = 0;
      *Z_ptr(2, z, y, x, n, d) = 0;
    }
  }
}

template <typename T>
__global__
void kernel_copy_Oxp_to_Q(const int H, const int W, const int N, const int D,
                          const T* O, T* Q) {
  for (int ii = thGi; ii < 4 * H * W * N * D; ii += NTG) {
    const int d = ii % D;
    const int n = (ii / D) % N;
    const int x = (ii / (N * D)) % W;
    const int y = (ii / (W * N * D)) % H;
    const int z = ii / (H * W * N * D);
    const int xp = (z == 0 || z == 2) ? x - 1 : x + 1; // previous x
    Q[(z * H * W * N * D) + (y * W * N * D) + (x * N * D) + (n * D) + d] =
        xp >= 0 && xp < W ? *O_ptr(y, xp, n, z, d) : 0;
  }
}

template <typename T>
__global__
void kernel_copy_Oyp_to_Q(const int H, const int W, const int N, const int D,
                          const T* O, T* Q) {
  for (int ii = thGi; ii < 4 * H * W * N * D; ii += NTG) {
    const int d = ii % D;
    const int n = (ii / D) % N;
    const int x = (ii / (N * D)) % W;
    const int y = (ii / (W * N * D)) % H;
    const int z = ii / (H * W * N * D);
    const int yp = (z == 0 || z == 1) ? y - 1 : y + 1;  // previous y
    Q[(z * H * W * N * D) + (y * W * N * D) + (x * N * D) + (n * D) + d] =
        yp >= 0 && yp < H ? *O_ptr(yp, x, n, z, d) : 0;
  }
}

template <typename T>
__global__
void kernel_copy_dO_to_Z(const int H, const int W, const int N, const int D,
                         const T* dO, T* Z) {
  for (int ii = thGi; ii < H * W * N * 4 * D; ii += NTG) {
    const int d = ii % D;                    // d \in [0 ... D-1]
    const int z = (ii / D) % 4;              // z \in [0 ... 3]
    const int n = (ii / (4 * D)) % N;        // n \in [0 ... N-1]
    const int x = (ii / (N * 4 * D)) % W;    // x \in [0 ... W-1]
    const int y = (ii / (W * N * 4 * D));    // y \in [0 ... H-1]
    *Z_ptr(0, z, y, x, n, d) = *dO_ptr(y, x, n, z, d);
  }
}


/* 2D-LSTM backward pass running on the GPU
 * H -> maximum height
 * W -> maximum width
 * N -> batch size
 * K -> input dimensions/channels
 * D -> output dimensions/channels
 * I -> input data (layout: H x W x N x K)
 * S -> input sizes (height and width of each sample, layout: N x 2)
 * P -> parameters (size: 4 * (1 + K + D + D) * 5 * D)
 * O -> output data (layout: H x W x N x 4 x D)
 * Q -> gates pre-activations and cells (layout: 4 x H x W x N x 5 x D)
 * dO -> derivative of the loss w.r.t the output
 * dQ -> derivative of the loss w.r.t the internal states
 */
template <typename T, typename FG, typename FI, typename FO>
inline void bw_workspace(
    const int H, const int W, const int N, const int K, const int D,
    const T* I, const int* S, const T* P, const T* O, const T* dO,
    void* wspace, void* rspace) {
  CHECK_NOTNULL(I);
  CHECK_NOTNULL(P);
  CHECK_NOTNULL(O);
  CHECK_NOTNULL(dO);
  CHECK_NOTNULL(wspace);
  CHECK_NOTNULL(rspace);
  // Prepare cublas handler and streams
  hipblasHandle_t handle;
  CHECK_CUBLAS_CALL(hipblasCreate(&handle));

  T* Q = reinterpret_cast<T*>(rspace);
  T* Z = reinterpret_cast<T*>(wspace);

  // Copy errors from the next layer(s) to the workspace
  kernel_copy_dO_to_Z<T><<<GRID_SIZE, BLOCK_SIZE>>>(H, W, N, D, dO, Z);
  CHECK_LAST_CUDA_CALL();

  const T** ptrs_gpu = reinterpret_cast<const T**>(
      (char*)wspace + sizeof(T) * (3 * 4 * H * W * N * D));
  const T** ptrs_cpu = nullptr;
  CHECK_CUDA_CALL(hipHostMalloc(
      &ptrs_cpu, sizeof(const T**) * 2 * 3 * 4 * std::min(H, W)));

  // Process the image diagonal-wise, in backwards order (there are H + W - 1
  // diagonals to process)
  {
    const T** ptrs_cpu_x = ptrs_cpu + 0 * 3 * 4 * std::min(H, W);
    const T** ptrs_cpu_y = ptrs_cpu + 1 * 3 * 4 * std::min(H, W);

    for (int t = H + W - 2; t >= 0; --t) {
      // Compute number of elements in the diagonal
      const int Tmin = std::max(0, t - W + 1);
      const int Tmax = std::min(t, H - 1);
      const int Tn   = (Tmax - Tmin) + 1;

      // Matrix multiplications to compute dJ/dO(x,y).
      // Notice that the loss function is not only affected by the output
      // at time (x,y) but also at times (x+1,y) and (x,y+1)!
      int batch_mul_size_x = 0, batch_mul_size_y = 0;
      for (int z = 0; z < 4; ++z) {
        for (int e = 0; e < Tn; ++e) {
          const int i = e + Tmin;
          const int j = t - i;
          const int y  = (z == 0 || z == 1) ? i : H - i - 1;
          const int x  = (z == 0 || z == 2) ? j : W - j - 1;
          const int yn = (z == 0 || z == 1) ? y + 1 : y - 1;  // next y
          const int xn = (z == 0 || z == 2) ? x + 1 : x - 1;  // next x
          if (yn >= 0 && yn < H) {
            ptrs_cpu_y[batch_mul_size_y + 0 * 4 * std::min(H, W)] =
                Q_ptr(z, yn, x, 0, 0, 0);
            ptrs_cpu_y[batch_mul_size_y + 1 * 4 * std::min(H, W)] =
                U_ptr(z, 0, 0, 0);
            ptrs_cpu_y[batch_mul_size_y + 2 * 4 * std::min(H, W)] =
                Z_ptr(0, z, y, x, 0, 0);
            ++batch_mul_size_y;
          }
          if (xn >= 0 && xn < W) {
            ptrs_cpu_x[batch_mul_size_x + 0 * 4 * std::min(H, W)] =
                Q_ptr(z, y, xn, 0, 0, 0);
            ptrs_cpu_x[batch_mul_size_x + 1 * 4 * std::min(H, W)] =
                V_ptr(z, 0, 0, 0);
            ptrs_cpu_x[batch_mul_size_x + 2 * 4 * std::min(H, W)] =
                Z_ptr(0, z, y, x, 0, 0);
            ++batch_mul_size_x;
          }
        }
      }

      // Copy pointers to the gpu for batched multiplications
      CHECK_CUDA_CALL(
          hipMemcpy(ptrs_gpu, ptrs_cpu,
                     sizeof(const T**) * 2 * 3 * 4 * std::min(H, W),
                     hipMemcpyHostToDevice));
      // [A,Gi,Go,Gx,Gy](x,y) += O(x-1,y) * [V_a,V_i,V_o,V_x,V_y]
      const T** dQx_ptrs = ptrs_gpu + 0 * 4 * std::min(H, W);
      const T** V_ptrs = ptrs_gpu + 1 * 4 * std::min(H, W);
      T** Zx_ptrs = const_cast<T**>(ptrs_gpu) + 2 * 4 * std::min(H, W);
      CHECK_CUBLAS_CALL(
          gemm_gpu_batched<T>(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, D, 5 * D,
                              1.0, dQx_ptrs, 5 * D, V_ptrs, 5 * D,
                              1.0, Zx_ptrs, D, batch_mul_size_x));
      // [A,Gi,Go,Gx,Gy](x,y) += O(x,y-1) * [U_a,U_i,U_o,U_x,U_y]
      const T** dQy_ptrs = ptrs_gpu + (3 + 0) * 4 * std::min(H, W);
      const T** U_ptrs = ptrs_gpu + (3 + 1) * 4 * std::min(H, W);
      T** Zy_ptrs = const_cast<T**>(ptrs_gpu + (3 + 2) * 4 * std::min(H, W));
      CHECK_CUBLAS_CALL(
          gemm_gpu_batched<T>(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, D, 5 * D,
                              1.0, dQy_ptrs, 5 * D, U_ptrs, 5 * D,
                              1.0, Zy_ptrs, D, batch_mul_size_y));
      // Compute cell and output values:
      // C(x, y) = f(A(x,y)) * f(Gi(x,y)) +
      //           C(x,y-1)  * f(Gy(x,y))  +
      //           C(x-1,y)  * f(Gx(x,y))
      // O(x, y) = f(C(x,y)) * f(Go(x,y))
      kernel_bw_elemwise_ops<T, FG, FI, FO>
          <<<GRID_SIZE, BLOCK_SIZE>>>(H, W, N, D, t, Tn, Tmin, S, Q, Z);
      CHECK_LAST_CUDA_CALL();
    }
  }

  CHECK_CUBLAS_CALL(hipblasDestroy(handle));
  CHECK_CUDA_CALL(hipHostFree(ptrs_cpu));
}

template <typename T>
inline void bw_input(
    const int H, const int W, const int N, const int K, const int D,
    const T* P, const T scale, T* dI, void *wspace, void* rspace) {
  CHECK_NOTNULL(P);
  CHECK_NOTNULL(dI);
  CHECK_NOTNULL(wspace);
  CHECK_NOTNULL(rspace);
  hipblasHandle_t handle;
  CHECK_CUBLAS_CALL(hipblasCreate(&handle));
  T* Q = reinterpret_cast<T*>(rspace);

  // Compute dJ/dI(y,x)
  for (int z = 0; z < 4; ++z) {
    CHECK_CUBLAS_CALL(gemm_gpu<T>(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_T, H * W * N, K, 5 * D,
        scale, Q_ptr(z, 0, 0, 0, 0, 0), 5 * D,
        W_ptr(z, 0, 0, 0), 5 * D,
        1.0, dI, K));
  }
  CHECK_CUBLAS_CALL(hipblasDestroy(handle));
}

template <typename T>
inline void bw_param(
    const int H, const int W, const int N, const int K, const int D,
    const T* I, const T* O, const T scale, T* dP, void* wspace, void* rspace) {
  CHECK_NOTNULL(I);
  CHECK_NOTNULL(O);
  CHECK_NOTNULL(dP);
  CHECK_NOTNULL(wspace);
  CHECK_NOTNULL(rspace);

  T* Q = reinterpret_cast<T*>(rspace);
  T* Z = reinterpret_cast<T*>(wspace);

  hipblasHandle_t handle;
  CHECK_CUBLAS_CALL(hipblasCreate(&handle));
  hipStream_t stream[4 * 4];
  STREAMS_CREATE(4 * 4);

  // dJ/db
  T* vOnes = Z;
  kernel_fill1D<T><<<GRID_SIZE, BLOCK_SIZE>>>(H * W * N, vOnes, 1);
  CHECK_LAST_CUDA_CALL();
  for (int z = 0; z < 4; ++z) {
    CHECK_CUBLAS_CALL(hipblasSetStream(handle, stream[z]));
    CHECK_CUBLAS_CALL(gemv_gpu<T>(
        handle, HIPBLAS_OP_T, H * W * N, 5 * D,
        scale, Q_ptr(z, 0, 0, 0, 0, 0), 5 * D, vOnes, 1,
        1.0, dB_ptr(z, 0, 0), 1));
  }

  // dJ/dW
  for (int z = 0; z < 4; ++z) {
    CHECK_CUBLAS_CALL(hipblasSetStream(handle, stream[4 + z]));
    CHECK_CUBLAS_CALL(gemm_gpu<T>(
        handle, HIPBLAS_OP_T, HIPBLAS_OP_N, K, 5 * D, H * W * N,
        scale, I, K, Q_ptr(z, 0, 0, 0, 0, 0), 5 * D,
        1.0, dW_ptr(z, 0, 0, 0), 5 * D));
  }

  // translate the output tensor in the x-dimension
  T* Oxp = Z + H * W * N;
  kernel_copy_Oxp_to_Q
      <<<GRID_SIZE, BLOCK_SIZE, 0, stream[8]>>>(H, W, N, D, O, Oxp);
  // translate the output tensor in the y-dimension
  T* Oyp = Z + H * W * N + 4 * H * W * N * D;
  kernel_copy_Oyp_to_Q
      <<<GRID_SIZE, BLOCK_SIZE, 0, stream[9]>>>(H, W, N, D, O, Oyp);
  // wait for data copies
  CHECK_CUDA_CALL(hipStreamSynchronize(stream[8]));
  CHECK_CUDA_CALL(hipStreamSynchronize(stream[9]));

  // dJ/dRx
  for (int z = 0; z < 4; ++z) {
    CHECK_CUBLAS_CALL(hipblasSetStream(handle, stream[8 + z]));
    CHECK_CUBLAS_CALL(gemm_gpu<T>(
        handle, HIPBLAS_OP_T, HIPBLAS_OP_N, D, 5 * D, H * W * N,
        scale, Oxp + z * H * W * N * D, D,
        Q_ptr(z, 0, 0, 0, 0, 0), 5 * D,
        1.0, dV_ptr(z, 0, 0, 0), 5 * D));
  }

  // dJ/dRy
  for (int z = 0; z < 4; ++z) {
    CHECK_CUBLAS_CALL(hipblasSetStream(handle, stream[12 + z]));
    CHECK_CUBLAS_CALL(gemm_gpu<T>(
        handle, HIPBLAS_OP_T, HIPBLAS_OP_N, D, 5 * D, H * W * N,
        scale, Oyp + z * H * W * N * D, D,
        Q_ptr(z, 0, 0, 0, 0, 0), 5 * D,
        1.0, dU_ptr(z, 0, 0, 0), 5 * D));
  }

  STREAMS_SYNCHRONIZE(4 * 4);
  STREAMS_DESTROY(4 * 4);
  CHECK_CUBLAS_CALL(hipblasDestroy(handle));
}



#define DEFINE_WRAPPERS(DEVICE, TYPE)                                   \
  void rnn2d_lstm_ ## DEVICE ## _ ## TYPE ## _fw_inference(             \
      const int H, const int W, const int N, const int K, const int D,  \
      const TYPE* input, const int* shape, const TYPE* param,           \
      TYPE* output, void* workspace) {                                  \
    fw_training< TYPE, Sigmoid<TYPE>, Tanh<TYPE>, Tanh<TYPE> >(         \
        H, W, N, K, D, input, shape, param, output,                     \
        workspace, nullptr);                                            \
  }                                                                     \
                                                                        \
  void rnn2d_lstm_ ## DEVICE ## _ ## TYPE ## _fw_training(              \
      const int H, const int W, const int N, const int K, const int D,  \
      const TYPE* input, const int* shape, const TYPE* param,           \
      TYPE* output, void* workspace, void* reserve) {                   \
    fw_training< TYPE, Sigmoid<TYPE>, Tanh<TYPE>, Tanh<TYPE> >(         \
        H, W, N, K, D, input, shape, param, output,                     \
        workspace, reserve);                                            \
  }                                                                     \
                                                                        \
  void rnn2d_lstm_ ## DEVICE ## _ ## TYPE ## _bw_workspace(             \
      const int H, const int W, const int N, const int K, const int D,  \
      const TYPE* input, const int* shape, const TYPE* param,           \
      const TYPE* output, const TYPE* dOutput,                          \
      void* workspace, void* reserve) {                                 \
    bw_workspace< TYPE, Sigmoid<TYPE>, Tanh<TYPE>, Tanh<TYPE> >(        \
        H, W, N, K, D, input, shape, param, output, dOutput,            \
        workspace, reserve);                                            \
  }                                                                     \
                                                                        \
  void rnn2d_lstm_ ## DEVICE ## _ ## TYPE ## _bw_input(                 \
      const int H, const int W, const int N, const int K, const int D,  \
      const TYPE* param, const TYPE scale, TYPE* dInput,                \
      void* workspace, void* reserve) {                                 \
    bw_input< TYPE >(H, W, N, K, D, param, scale, dInput,               \
                     workspace, reserve);                               \
  }                                                                     \
                                                                        \
  void rnn2d_lstm_ ## DEVICE ## _ ## TYPE ## _bw_param(                 \
      const int H, const int W, const int N, const int K, const int D,  \
      const TYPE* input, const TYPE* output, const TYPE scale,          \
      TYPE* dParam, void* workspace, void* reserve) {                   \
    bw_param< TYPE >(H, W, N, K, D, input, output, scale, dParam,       \
                     workspace, reserve);                               \
  }                                                                     \
                                                                        \
  size_t rnn2d_lstm_ ## DEVICE ## _ ## TYPE ## _inference_workspace_size( \
      const int H, const int W, const int N, const int D) {             \
    return get_inference_workspace_size<TYPE>(H, W, N, D);              \
  }                                                                     \
                                                                        \
  size_t rnn2d_lstm_ ## DEVICE ## _ ## TYPE ## _training_workspace_size( \
      const int H, const int W, const int N, const int D) {             \
    return get_training_workspace_size<TYPE>(H, W, N, D);               \
  }                                                                     \
                                                                        \
  size_t rnn2d_lstm_ ## DEVICE ## _ ## TYPE ## _training_reserve_size(  \
      const int H, const int W, const int N, const int D) {             \
    return get_training_reserve_size<TYPE>(H, W, N, D);                 \
  }

extern "C" {
  DEFINE_WRAPPERS(gpu, float)
  DEFINE_WRAPPERS(gpu, double)
}  // extern "C"
